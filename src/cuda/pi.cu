
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <array>
#include <iostream>

inline constexpr size_t kWarpSize = 32;
inline constexpr size_t kCudaCores = 2048;
inline constexpr size_t kIterations = 1000000;

__global__ auto piCount(uint64_t* totals) -> void {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  totals[idx] = 0;
  hiprandState state;
  hiprand_init(1234, idx, 0, &state);
  for (size_t i = 0; i < kIterations; i++) {
    float x = hiprand_uniform(&state);
    float y = hiprand_uniform(&state);
    totals[idx] += 1 - static_cast<uint64_t>(x * x + y * y);
  }
}

auto main() -> int {
  constexpr size_t N = kCudaCores * kWarpSize;
  uint64_t* counts;
  hipMalloc(&counts, N * sizeof(uint64_t));
  piCount<<<kCudaCores, kWarpSize>>>(counts);

  std::array<uint64_t, N> hostCounts;
  hipMemcpy(hostCounts.data(), counts, N * sizeof(uint64_t), hipMemcpyDeviceToHost);
  hipFree(counts);
  
  uint64_t total = 0;
  for (const uint64_t count : hostCounts) {
    total += count;
  }
  const double pi = 4.0 * static_cast<double>(total) / static_cast<double>(kIterations * kCudaCores * kWarpSize);
  std::cout << "Pi: " << pi << std::endl;
  return 0;
}

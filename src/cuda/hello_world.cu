#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ auto helloCUDA() -> void {
  printf("Hello CUDA from GPU!\n");
}

auto main() -> int {
  helloCUDA<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}

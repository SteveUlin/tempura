#include <cstdio>
#include <hip/hip_runtime.h>

__global__ auto helloCUDA() -> void {
  printf("Hello CUDA from GPU!\n");
}

auto main() -> int {
  auto a = [] {
    helloCUDA<<<1, 1>>>();
  };
  hipDeviceSynchronize();
  return 0;
}

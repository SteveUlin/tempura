
#include <hip/hip_runtime.h>
#include <array>
#include <numeric>
#include <iostream>

__global__ auto VecAdd10(float* A, int N) -> void {
  const size_t i  = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    A[i] = A[i] + 10;
  }
}

auto main() -> int {
  constexpr size_t N = 10;
  std::array<float, N> arr;
  std::iota(arr.begin(), arr.end(), 0);

  float* d_arr;
  hipMalloc(&d_arr, N * sizeof(float));
  hipMemcpy(d_arr, arr.data(), N * sizeof(float), hipMemcpyHostToDevice);

  VecAdd10<<<1, N>>>(d_arr, N);

  hipMemcpy(arr.data(), d_arr, N * sizeof(float), hipMemcpyDeviceToHost);

  for (const float x : arr) {
    std::cout << x << std::endl;
  }
  return 0;
}

